#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "cudaUtils.h"
#include "kernel.h"

hipError_t FreeFunction(double * dev_W, double * dev_alfa, int * dev_mislead, int * dev_tempresult)
{
	hipError_t cudaStatus;
	cudaStatus = hipFree(dev_W);
	if (cudaStatus != hipSuccess) {

		printf("failed to free cuda - W  \n");
	}
	cudaStatus = hipFree(dev_mislead);
	if (cudaStatus != hipSuccess) {

		printf("failed to free cuda - mislead points \n");
	}

	cudaStatus = hipFree(dev_tempresult);
	if (cudaStatus != hipSuccess) {

		printf("failed to free cuda - tempresult \n");
	}

	cudaStatus = hipFree(dev_alfa);
	if (cudaStatus != hipSuccess) {

		printf("failed to free cuda - alfa \n");
	}

	return cudaStatus;
}

void MyCudaMalloc(void** dev_pointer, size_t size, int error_label)
{
	hipError_t cudaStatus;

	// points malloc n dims  .
	cudaStatus = hipMalloc(dev_pointer, size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed! error_label : %d ", error_label);

		MyCudaFree(*dev_pointer, error_label);
	}


}

void MyCudaCopy(void* dest, void * src, size_t size, hipMemcpyKind kind, int error_label)
{
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(dest, src, size, kind);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! error_label : %d", error_label);
	}
}

void MyCudaFree(void * object, int error_label)
{
	hipError_t cudaStatus;
	cudaStatus = hipFree(object);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! error_label : %d", error_label);
	}
}

void FreeConstanstCuda(Point * dev_pts, double * dev_values, int * dev_n, int * dev_k)
{
	MyCudaFree(dev_pts, 03);
	MyCudaFree(dev_values, 04);
	MyCudaFree(dev_n, 05);
	MyCudaFree(dev_k, 06);

}

void mallocConstCuda(Point * pts, int n, int k, Point ** dev_pts, int ** dev_n, int ** dev_k, double ** dev_values)
{

	MyCudaMalloc((void**)&(*dev_pts), sizeof(Point)* n, 1);
	MyCudaMalloc((void**)&(*dev_values), sizeof(double)* (n*(k + 1)), 2); // value n * (k+1) each point have k+1 dims values
	MyCudaCopy((*dev_pts), pts, sizeof(Point)*n, hipMemcpyHostToDevice, 4);

	for (int i = 0; i < n; i++)
		MyCudaCopy(&(*dev_values)[i*(k + 1)], &pts[i].values[0], sizeof(double)*(k + 1), hipMemcpyHostToDevice, 5);


	MyCudaMalloc((void**)&(*dev_n), sizeof(int), 265);
	MyCudaMalloc((void**)&(*dev_k), sizeof(int), 3);
	MyCudaCopy((*dev_n), &n, 1, hipMemcpyHostToDevice, 5);
	MyCudaCopy((*dev_k), &k, 1, hipMemcpyHostToDevice, 6);


}

double ProcessAlfa(Point * dev_pts, double* dev_values, double  * alfa, int *dev_n,
	int *dev_k, int limit, double QC, int n, int k, double ** WSaved)
{
	*WSaved = (double*)malloc((k + 1) * sizeof(double)); // W k+1 dims 
	int * tempresult = (int*)malloc(n * sizeof(int)); // temp result will collect mislead points for q
	int * mislead = (int*)malloc(n * sizeof(int)); // array of n points , 
	//mislead points will be 1 or -1 ,currect=0
	int * dev_mislead = NULL;
	double * dev_W = NULL;
	double * dev_alfa = NULL;
	int * dev_tempresult = NULL;
	hipError_t cudaStatus;

#pragma region malloc and copy values to GPU


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);

		return -2;
	}

	// w , mislead_pts , dev_alfa,tempresult,


	MyCudaMalloc((void**)&dev_W, sizeof(double)* (k + 1), 7);
	hipMemset(dev_W, 0, sizeof(double)* (k + 1));

	//MyCudaCopy(dev_W, W, sizeof(double)*(k + 1), hipMemcpyHostToDevice, 8);

	MyCudaMalloc((void**)&dev_mislead, sizeof(int)* (n), 9);
	hipMemset(dev_mislead, 0, sizeof(int)* (n));

	//MyCudaCopy(dev_mislead, mislead, sizeof(int)*(n), hipMemcpyHostToDevice, 10);

	MyCudaMalloc((void**)&dev_alfa, sizeof(double), 11);
	MyCudaCopy(dev_alfa, alfa, sizeof(double), hipMemcpyHostToDevice, 12);

	MyCudaMalloc((void**)&dev_tempresult, sizeof(int)*n, 13);
	MyCudaCopy(dev_tempresult, tempresult, sizeof(int)*n, hipMemcpyHostToDevice, 14);
	hipMemset(dev_tempresult, 0, sizeof(int)* (n));

	MyCudaCopy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice, 14);




	//// TODO get all values from devices;
	//Point * pts2 = (Point*)malloc(sizeof(Point)*n);


	//MyCudaCopy(W,dev_W, sizeof(double)*(k + 1), hipMemcpyDeviceToHost, 100);
	//MyCudaCopy(mislead, dev_mislead, sizeof(int)*(n), hipMemcpyDeviceToHost, 101);
	//MyCudaCopy(alfa, dev_alfa, sizeof(double), hipMemcpyDeviceToHost, 120);
	//MyCudaCopy(tempresult, dev_tempresult, sizeof(int)*n, hipMemcpyDeviceToHost, 140);

	//MyCudaCopy(pts2, dev_pts, sizeof(int)*n, hipMemcpyDeviceToHost, 145);

	//MyCudaCopy(&k, dev_k, sizeof(int), hipMemcpyDeviceToHost, 14);
	//
	//	printf("i=%d0 , cuda w : %lf \n",0, W[0]);
	//	printf(" cuda alfa : %lf \n", *alfa);
	//	printf("i=%d,cuda mislead : %d \n",0, mislead[0]);
	//	printf("i=%d,cuda tempresult : %d \n",0, tempresult[0]); // WORKS
	//printf("cuda n = %d  \n", n);
	//

	//printf("point 0  = (%f,%f,%f,%f)  group = %d \n", pts2[3].values[0], pts2[3].values[1], pts2[3].values[2] ,pts2[3].values[3],pts2[3].group);




#pragma  endregion


	int threadDims = 1000;
	int blockDims = (n / threadDims) + 1;
	int counter_limit = 0;

	while (counter_limit < limit)
	{
		// get all mislead points
		getMisLeadArrayFromPoints << <blockDims, threadDims >> > (dev_pts, dev_values, dev_W, dev_mislead, dev_k, dev_n);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "loopOverPoints2 launch failed: %s\n", hipGetErrorString(cudaStatus));
			FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
			return -2;

		}
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching loopOverPoints! : %s \n", cudaStatus, hipGetErrorString(cudaStatus));
			FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
			return -2;

		}

		MyCudaCopy(mislead, dev_mislead, (n) * sizeof(int), hipMemcpyDeviceToHost, 15);


		int indexerMiss = 0;
		int result;
		// we get a array of mislead values (0 == ok , else (1,-1) false));
		for (indexerMiss = 0; indexerMiss < n; indexerMiss++)
		{
			result = mislead[indexerMiss];
			if (result == -1 || result == 1) // found point that mislead
				break;
		}

		if (indexerMiss == n)// all point in good places
			break;
		else
		{
			// need to create a new W 

			*alfa = *alfa*mislead[indexerMiss]; // alfa * sign
			MyCudaCopy(dev_alfa, alfa, sizeof(double), hipMemcpyHostToDevice, 77);

			*alfa = fabs(*alfa); // back to postive alfa

			int indexValues = indexerMiss * (k + 1);
			int * dev_index_values = NULL;
			MyCudaMalloc((void**)&dev_index_values, sizeof(int), 88);
			MyCudaCopy(dev_index_values, &indexValues, sizeof(int), hipMemcpyHostToDevice, 99);
			

			//create a new weight
			createNewWeight << <1, k + 1 >> > (dev_alfa, dev_values, dev_index_values, dev_W);

			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "createNewWeight launch failed: %s\n", hipGetErrorString(cudaStatus));
				FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
				return -2;

			}
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching createNewWeight! : %s \n", cudaStatus, hipGetErrorString(cudaStatus));
				FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
				return -2;

			}
		}
		counter_limit++;
	}

	// need to calcate the q , get all mislead point  , to dev_tempresult
	getMisLeadArrayFromPoints << <blockDims, threadDims >> > (dev_pts, dev_values, dev_W, dev_tempresult, dev_k, dev_n);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "getMisLeadArrayFromPoints Second Time launch failed: %s\n", hipGetErrorString(cudaStatus));
		FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
		return -2;

	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching getMisLeadArrayFromPoints Second Time! : %s \n", cudaStatus, hipGetErrorString(cudaStatus));
		FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);
		return -2;

	}

	MyCudaCopy(tempresult, dev_tempresult, (n) * sizeof(int), hipMemcpyDeviceToHost, 15);

	int sumOFmisLead = 0;
	for (int i = 0; i < n; i++)
	{
		if (tempresult[i] != 0)
			sumOFmisLead += 1;
	}

	double q = sumOFmisLead / (n*(1.0));
	MyCudaCopy(*WSaved, dev_W, sizeof(double)*(k + 1), hipMemcpyDeviceToHost, 70); // copy W

	// clear resources on cuda GPU
	FreeFunction(dev_W, dev_alfa, dev_mislead, dev_tempresult);

	if (q <= QC)
		return q;
	else
		return 2.0; // q that never will get and larger from all q possiblies .



}
